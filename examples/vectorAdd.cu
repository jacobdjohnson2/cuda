#include <iostream>
#include <hip/hip_runtime.h>

#define SIZE 1024

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = threadIdx.x;
    // int i = blockIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int *a, *c;
    hipMallocManaged(&a, SIZE * sizeof(int));
    hipMallocManaged(&c, SIZE * sizeof(int));

    for (auto i = 0; i < SIZE; ++i) {
        a[i] = i;
        c[i] = 0;
    }

    vectorAdd<<<1, SIZE>>>(a, a, c, SIZE);
    hipDeviceSynchronize();

    for (auto i = 0; i < 10; ++i) {
        std::cout << c[i] << std::endl;
    }

    hipFree(a);
    hipFree(c);
    
    return 0;
}